#include "hip/hip_runtime.h"
//LOS CODIGOS FUERON MODIFICADOS DEL MATERIAL DADOD EN CLASE CON EL OBJETIVO DE QUE FUERA MAS FACIL COMPRENDER
//SE USARON ALGUNAS PAGINAS DE INTERNET Y LIBROS Y CODIGO COMENTADO, LAS REFENECIAS ESTAN EN EL REPORTE Y EN CODIGO

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <chrono>
#include "common.h"

using namespace std;

/*
__global__ void matrixMultOnHostGPU(int *a, int *b, int *c) {
 int k, sum = 0;
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int fil = threadIdx.y + blockDim.y * blockIdx.y;

 if (col < N && fil < N) {
  for (k = 0; k < N; k++) {
   sum += a[fil * N + k] * b[k * N + col];
  }
  c[fil * N + col] = sum;
 }
}*/
//Multiplicacion en GPU
__global__ void matrixMultOnHostGPU1D(long *MatA, long *MatB, long *MatC, const int N)
{
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = blockIdx.y + blockIdx.y * blockDim.y;
  //verificacion de las filas para la multiplicacion
  if (ix < N && iy < N){
    for (int k = 0; k < N; k++){
      //sum += a[fil * N + k] * b[k * N + col];
      MatC[iy * N + ix] += MatA[iy * N + iy] * MatB[k * N +ix];
    }
  }
}

//Multiplicacion en CPU
void matrixMultOnHost(long * A, long * B, long * C, int N)
{
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      for (int k = 0; k < N; k++){
        //Operacion para hacer la regla del karatzo fila por culumna
        C[i * N + j] += A[i * N * k] * B[j + k * N];
      }
    }
  }
}

void checkResult(long *hostRef, long *gpuRef, const int N){
  double epsilon = 1.0E-8;
  bool match = 1;
  for (int i = 0; i < N; i++){
    if (abs(hostRef[i] - gpuRef[i]) > epsilon){
      match = 0;
      printf("host %ld gpu %ld\n", hostRef[i], gpuRef[i]);
      break;
    }
  }
  if (match)
    printf("Matrix match.\n");
  else
    printf("Matrix do not match.\n");
}


int main(int argc, char *argv[])
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // Tamaño de la matriz
    int N = 1000;
    int nBytes = N * N * sizeof(long);

    // host memory
    long *h_A = (long *)malloc(nBytes);
    long *h_B = (long *)malloc(nBytes);
    long *hostRef = (long *)malloc(nBytes);
    long *gpuRef = (long *)malloc(nBytes);

    // Matriz inicalizada
    for(int i = 0; i < N * N; i++ ) {
        h_A[i] = i+1;
        h_B[i] = i+1;
    }

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result SAFE_CALLs
    auto start_cpu = chrono::high_resolution_clock::now();
    matrixMultOnHost(h_A, h_B, hostRef, N);
    auto end_cpu = chrono::high_resolution_clock::now();

    chrono::duration<float, milli> duration_ms = end_cpu - start_cpu;
    printf("sumMatrixOnHost elapsed %f ms\n", duration_ms.count());

    // malloc device global memory
    long *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");
    SAFE_CALL(hipMemset(d_MatC, 0, nBytes), "");

    // invoke kernel at host side
    dim3 block(32, 32);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    printf("grid.x %d grid.y %d block.x %d block.y %d\n", grid.x, grid.y, block.x, block.y);
    //kernel
    start_cpu =  chrono::high_resolution_clock::now();
    matrixMultOnHostGPU1D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, N);
    hipDeviceSynchronize();
    end_cpu = std::chrono::high_resolution_clock::now();

    //Formula
    duration_ms = end_cpu - start_cpu;
    printf("Multiplicacionelapsed %f ms\n", duration_ms.count());


    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // Compare results
    checkResult(hostRef, gpuRef, N);

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return 0;
}
